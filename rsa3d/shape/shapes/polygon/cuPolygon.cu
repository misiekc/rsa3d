
#include <hip/hip_runtime.h>
/*
 * cuPolygon.cpp
 *
 *  Created on: 23.01.2019
 *      Author: ciesla
 */

#ifdef CUDA_ENABLED

#include <cuda_runtime.h>

#include "Polygon.h"

struct Pair{
	unsigned int first;
	unsigned int second;
};

float *Polygon::d_vertices = 0;
float *Polygon::d_angles = 0;
std::pair<unsigned int, unsigned int> *Polygon::d_segments = 0;
std::pair<unsigned int, unsigned int> *Polygon::d_helperSegments = 0;



	//test if line segment from point 1 to 2 intersects with line segment from point 3 to 4
__device__ bool cuLineLineIntersect(float x1, float y1, float x2, float y2, float x3, float y3, float x4, float y4){
	float o1 = (y2 - y1)*(x3 - x2) - (x2 - x1)*(y3 - y2);
	float o2 = (y2 - y1)*(x4 - x2) - (x2 - x1)*(y4 - y2);
	float o3 = (y4 - y3)*(x1 - x4) - (x4 - x3)*(y1 - y4);
	float o4 = (y4 - y3)*(x2 - x4) - (x4 - x3)*(y2 - y4);

	return (o1*o2 < 0.0) && (o3*o4 < 0.0);
}
/*
__global__ void cuLinesLinesIntersect(bool *results, float *firstSet, unsigned int firstSetLength, float *secondSet, unsigned int secondSetLength){
	// determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x; //threadIdx.x;

    unsigned int i = tid / secondSetLength;
    unsigned int j = tid - i*secondSetLength;

    if(i>=firstSetLength || j>=secondSetLength || i>=j)
    	return;

    float x1 = firstSet[4*i];
    float y1 = firstSet[4*i+1];
    float x2 = firstSet[4*i+2];
    float y2 = firstSet[4*i+3];
    float x3 = secondSet[4*j];
    float y3 = secondSet[4*j+1];
    float x4 = secondSet[4*j+2];
    float y4 = secondSet[4*j+3];

    results[tid] = cuLineLineIntersect(x1, y1, x2, y2, x3, y3, x4, y4);
}
*/
__global__ 	void cuPolygonPolygonOverlap(
		bool *results,
		float *vertices, float *angles,
		std::pair<unsigned int, unsigned int> *segments, unsigned int segmentsCount,
		float posX, float posY, float angle, float polposX, float polposY, float polangle){

	// determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x; //threadIdx.x;

    unsigned int i = tid / segmentsCount;
    unsigned int j = tid - i*segmentsCount;

    if (tid >= segmentsCount*segmentsCount)
    	return;

	float x1 = polposX + vertices[segments[i].first] * cos(angles[segments[i].first] + polangle);
	float y1 = polposY + vertices[segments[i].first] * sin(angles[segments[i].first] + polangle);
	float x2 = polposX + vertices[segments[i].second] * cos(angles[segments[i].second] + polangle);
	float y2 = polposY + vertices[segments[i].second] * sin(angles[segments[i].second] + polangle);

	float x3 = posX + vertices[segments[j].first] * cos(angles[segments[j].first] + angle);
	float y3 = posY + vertices[segments[j].first] * sin(angles[segments[j].first] + angle);
	float x4 = posX + vertices[segments[j].second] * cos(angles[segments[j].second] + angle);
	float y4 = posY + vertices[segments[j].second] * sin(angles[segments[j].second] + angle);

	results[tid] = cuLineLineIntersect(x1, y1, x2, y2, x3, y3, x4, y4);
}

__global__ 	void cuPolygonPolygonsOverlap(
		bool *results,
		float *vertices, float *angles,
		std::pair<unsigned int, unsigned int> *segments, unsigned int segmentsCount,
		float posX, float posY, float angle, float *polygons, unsigned int polygonsCount){

	// determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x; //threadIdx.x;

    unsigned int i = tid / (segmentsCount*segmentsCount); // number of polygon
    unsigned int j = tid / segmentsCount - i*segmentsCount;  // number of segment in the first polygon
    unsigned int k = tid - segmentsCount*(segmentsCount*i + j);  // number of segment in the second polygon

    if (tid >= segmentsCount*segmentsCount*polygonsCount)
    	return;

	float x1 = polygons[3*i]   + vertices[segments[j].first]  * cos(angles[segments[j].first]  + polygons[3*i+2]);
	float y1 = polygons[3*i+1] + vertices[segments[j].first]  * sin(angles[segments[j].first]  + polygons[3*i+2]);
	float x2 = polygons[3*i]   + vertices[segments[j].second] * cos(angles[segments[j].second] + polygons[3*i+2]);
	float y2 = polygons[3*i+1] + vertices[segments[j].second] * sin(angles[segments[j].second] + polygons[3*i+2]);

	float x3 = posX + vertices[segments[k].first]  * cos(angles[segments[k].first]  + angle);
	float y3 = posY + vertices[segments[k].first]  * sin(angles[segments[k].first]  + angle);
	float x4 = posX + vertices[segments[k].second] * cos(angles[segments[k].second] + angle);
	float y4 = posY + vertices[segments[k].second] * sin(angles[segments[k].second] + angle);

	results[tid] = cuLineLineIntersect(x1, y1, x2, y2, x3, y3, x4, y4);
}



void checkStatus(cudaError_t cudaResult, std::string msg){
    if (cudaResult != cudaSuccess){
        msg += cudaGetErrorString(cudaResult);
        printf("%s\n", msg.c_str());
        exit(0);
    }
}

/*
 * Copies polygon definition into device memory
 */
void Polygon::cuInit(){
    cudaError_t cudaResult = cudaSuccess;
    struct cudaDeviceProp deviceProperties;
    // Get device properties
    cudaResult = cudaGetDeviceProperties(&deviceProperties, 0);
    checkStatus(cudaResult, "Could not get device properties: ");

    // Attach to GPU
    cudaResult = cudaSetDevice(0);
    checkStatus(cudaResult, "Could not set device: ");

    printf("Device: %s\n", deviceProperties.name);
    printf("Block X size is %d\n", (unsigned int)deviceProperties.maxThreadsDim[0]);
    printf("Grid X size  is %d\n", (unsigned int)deviceProperties.maxGridSize[0]);

    unsigned int blockSize = (unsigned int)deviceProperties.maxThreadsDim[0];
    unsigned int size = Polygon::segments.size()+Polygon::helperSegments.size();
    unsigned int gridSize = (unsigned int) ( size*size / blockSize) + 1;


	float *vertices = new float[Polygon::vertexR.size()];
	for(size_t i=0; i<Polygon::vertexR.size(); i++){
		vertices[i] = (float)Polygon::vertexR[i];
	}
	cudaResult = cudaMalloc((void **)&Polygon::d_vertices, Polygon::vertexR.size() * sizeof(float));
    checkStatus(cudaResult, "Could not allocate memory on device for vertices: ");
	cudaResult = cudaMemcpy(Polygon::d_vertices, vertices, Polygon::vertexR.size()*sizeof(float), cudaMemcpyHostToDevice);
    checkStatus(cudaResult, "Could not copy vertices to device: ");
	delete[] vertices;

	float *angles = new float[Polygon::vertexTheta.size()];
	for(size_t i=0; i<Polygon::vertexTheta.size(); i++){
		angles[i] = (float)Polygon::vertexTheta[i];
	}
	cudaResult = cudaMalloc((void **)&Polygon::d_angles, Polygon::vertexTheta.size() * sizeof(float));
    checkStatus(cudaResult, "Could not allocate memory on device for angles: ");
	cudaResult = cudaMemcpy(Polygon::d_angles, angles, Polygon::vertexR.size()*sizeof(float), cudaMemcpyHostToDevice);
    checkStatus(cudaResult, "Could not copy angles to device: ");
	delete[] angles;

	std::pair<unsigned int, unsigned int> *segments = new std::pair<unsigned int, unsigned int>[Polygon::segments.size() + Polygon::helperSegments.size()];
	for(size_t i=0; i<Polygon::segments.size(); i++){
		segments[i] = Polygon::segments[i];
	}
	for(size_t i=0; i<Polygon::helperSegments.size(); i++){
		segments[Polygon::segments.size()+i] = Polygon::helperSegments[i];
	}
	cudaResult = cudaMalloc((void **)&Polygon::d_segments, (Polygon::segments.size()+Polygon::helperSegments.size()) * sizeof(std::pair<unsigned int, unsigned int>));
    checkStatus(cudaResult, "Could not allocate memory on device for segments: ");
	cudaResult = cudaMemcpy(Polygon::d_segments, segments, (Polygon::segments.size()+Polygon::helperSegments.size())*sizeof(std::pair<unsigned int, unsigned int>), cudaMemcpyHostToDevice);
    checkStatus(cudaResult, "Could not copy segments to device: ");
	delete[] segments;
}

void Polygon::cuFree(){
	cudaFree(Polygon::d_vertices);
	cudaFree(Polygon::d_angles);
	cudaFree(Polygon::d_segments);
	cudaFree(Polygon::d_helperSegments);
}

bool Polygon::overlap(BoundaryConditions<2> *bc, const Shape<2, 1> *s) const{
	Polygon pol = dynamic_cast<const Polygon&>(*s);
	this->applyBC(bc, &pol);

	float polposition[2];
	polposition[0] = (float)pol.getPosition()[0];
	polposition[1] = (float)pol.getPosition()[1];

	float position[2];
	position[0] = (float)this->getPosition()[0];
	position[1] = (float)this->getPosition()[1];

	//easy check
	double d2 = 0, tmp;
	for (unsigned short i = 0; i < 2; i++){
		tmp = position[i] - polposition[i];
		d2 += tmp*tmp;
	}
	if (std::sqrt(d2) < 2.0*Polygon::inscribedCircleRadius)
		return true;

	unsigned int size = Polygon::segments.size() + Polygon::helperSegments.size();
	cudaError_t cudaResult = cudaSuccess;

	bool *d_results = 0;
	cudaResult = cudaMalloc((void **)&d_results, size*size * sizeof(bool));
    checkStatus(cudaResult, "Could not allocate memory on device for results: ");

    cuPolygonPolygonOverlap<<<1, size*size>>>(
			d_results,
			Polygon::d_vertices, Polygon::d_angles,
			Polygon::d_segments, Polygon::segments.size(),
			position[0], position[1], this->getOrientation()[0], polposition[0], polposition[1], pol.getOrientation()[0]);

	bool *results = new bool[size*size];
	cudaResult = cudaMemcpy(results, d_results, size*size*sizeof(bool), cudaMemcpyDeviceToHost);
    checkStatus(cudaResult, "Could not copy results from device: ");
	cudaFree(d_results);

	for(int i=0; i<size; i++)
		for(int j=0; j<size; j++)
			if (results[i*size + j]){
				delete[] results;
				return true;
			}
	delete[] results;
	return false;
}

const Shape<2,1> * Polygon::overlap(BoundaryConditions<2> *bc, std::vector<const Shape<2, 1> *> *shapes) const{
	if (shapes->size()==0){
		return nullptr;
	}
	float *polygons = new float[3*shapes->size()];
	size_t i = 0;
	for(const Shape<2, 1> *s: *shapes){
		Polygon pol = dynamic_cast<const Polygon&>(*s);
		this->applyBC(bc, &pol);

		polygons[3*i] = (float)pol.getPosition()[0];
		polygons[3*i+1] = (float)pol.getPosition()[1];
		polygons[3*i+2] = (float)pol.getOrientation()[0];
		i++;
	}

	cudaError_t cudaResult = cudaSuccess;

	float *d_polygons = 0;
	cudaResult = cudaMalloc((void **)&d_polygons, 3*shapes->size() * sizeof(float));
    checkStatus(cudaResult, "Could not allocate memory on device for polygons: ");
	cudaResult = cudaMemcpy(d_polygons, polygons, 3*shapes->size() * sizeof(float), cudaMemcpyHostToDevice);
    checkStatus(cudaResult, "Could not copy polygons to device: ");
	delete[] polygons;

	unsigned int size = Polygon::segments.size() + Polygon::helperSegments.size();
	bool *d_results = 0;
	cudaResult = cudaMalloc((void **)&d_results, shapes->size()*size*size*sizeof(bool));
    checkStatus(cudaResult, "Could not allocate memory on device for results: ");

    cuPolygonPolygonsOverlap<<<1, size*size*shapes->size()>>>(
			d_results,
			Polygon::d_vertices, Polygon::d_angles,
			Polygon::d_segments, Polygon::segments.size(),
			this->getPosition()[0], this->getPosition()[1], this->getOrientation()[0], d_polygons, shapes->size());

	bool *results = new bool[size*size*shapes->size()];
	cudaResult = cudaMemcpy(results, d_results, size*size*shapes->size()*sizeof(bool), cudaMemcpyDeviceToHost);
    checkStatus(cudaResult, "Could not copy results from device: ");
	cudaFree(d_results);
	cudaFree(d_polygons);

	for(unsigned int i=0; i<shapes->size(); i++)
		for(unsigned int j=0; j<size; j++)
			for(unsigned int k=0; k<size; k++)
				if (results[i*size*size + j*size + k]){
					delete[] results;
					return (*shapes)[i];
				}
	delete[] results;
	return nullptr;
}
#endif
